#include "hip/hip_runtime.h"
# Single-line comments start with #

"Strings can be used as comments"

"""
Multiline strings are 
made using triple quotes.
"""

# Contants can be defined with the define statement 
define DAYS_IN_YEAR 365

# The value following the define keyword can also be optional 
# and used by the C preprocessor
define MACRO_VARIABLE

# Enumerations can be defined as a list of keywords.
# Unlike C/C++ enums, default values cannot be set in these.
enum days {SUN, MON, TUE, WED, THU, FRI, SAT}

# Import local headers with include 
# NOTE: System level includes are not required because all functions, 
# variables, and types defined in the C standard headers are 
# treated as builtin functions.
include "myheader.hu"

# Function declarations are similar to function definitions in python,
# but with type annotations and no body.
# These should generally be declared in your header files.
def function_1() -> void 
def demo_func_ptrs() -> void

# Function arguments can be specified with or without a type.
# NOTE: If a type is not specified, the type is inferred based on the types 
# of variables passed to this function as arguments when it is called.
def add_two_ints(x1: int, x2: int) -> int
def add_two_things(x1: int, x2: int) -> int

# NOTE: Functions that are declared without a type automatically default
# to int type, like C.
def func_no_specified_ret()

# The program's entry poiny is a function called "main"
# which can accept either no arguments, or 2 arguments, and 
# returns a string, similar to C.
def main(argc: int, argv: char[][]):
    # Also could have been declared as:
    # def main()
    # def main(argc, argv)  # type inference fills in the types

    pass  # Just a regular no-op 

    # printf() is the same as printf in c.
    # stdio.h does not have to be included b/c all functions defined 
    # in c standard headers are included as builtin functions.
    printf("%d\n", 0)

    """
    Types
    """

    # Variables can initially be declared with or without the type 
    x: int  # x declared as an int
    y: int = 4  # y declared as an int defaulted to 4
    z = 4  # First declaration of z, which is declared as the type of 
           # the right hand side of the assignment 

    # chars are 1 byte 
    x_char: char = 0
    y_char = 'y'  # Char literals are quoted with single quotes

    # shorts are usually 2 bytes 
    x_short: short 

    # ints are usually 4 bytes
    x_int: int 

    # longs are compiled to long longs in C and usually about 64 bits 
    x_long: long 

    # The previous types are all signed.
    # Unsigned versions of the previously mentioned types start with a 'u'
    x_uint: uint 

    # floats are usually 32-bit floating point numbers
    x_float: float = 0.0

    # doubles are usually 64-bit floating point numbers
    x_double: double = 0.0

    # The sizeof function will give you the the size of a variable or type
    # in bytes. Only the function is available, not the sizeof statement.
    int_size = sizeof(int)
    large_num_size = sizeof(9223372036854775807)  # 2^63
    printf("sizeof(int): %zu\n", int_size)
    printf("sizeof(2^63): %zu\n", large_num_size)

    # Just like in C, if the sizeof function is called on an expression,
    # that expression is not evaluated.
    a = 1 
    printf("original value of a: %d\n", a)
    size = sizeof(a++)
    printf("sizeof(a++) = %zu where new value of a = %d\n", size, a)

    # Arrays must be initialized using a concrete size 
    my_char_array: char[20]  # This array occupies 20 bytes
    my_int_array: int[20]  # This array occupies 80 bytes

    # You can initialize an array to all zeros thusly
    my_array: char[20] = [0]

    # If the type is not specified for an array literal,
    # it defaults to an int array of the literal's size
    my_array_default = [1, 2, 3]  # int array of size 3

    # Index an array just like any language 
    my_array[0]

    # Arrays are mutable 
    my_array[1] = 2 
    printf("%d\n", my_array[1])

    # Array sizes do not need to be declared at compile time. 
    # These are variable-lenght arrays.
    printf("Enter the array size: ")
    array_size: int 
    fscanf(stdin, "%d", &array_size)
    var_length_array: int[array_size]
    printf("sizeof(var_length_array) in bytes: %zu\n", sizeof(var_length_array))

    # Strings are arrays of chars terminated by a NULL character (0x00)
    # String literals already include the null character bu default.
    a_string = "This is a string"
    printf("%s\n", a_string)
    printf("%d\n", a_string[16])

    # Multidimensional arrays 
    multi_array = [
        [1, 2, 3, 4, 5],
        [6, 7, 8, 9, 10]
    ]

    # Access elements 
    array_int = multi_array[0][2]
    printf("%d\n", array_int)  # 3 
    assert(9 == multi_array[1][2] + 1)

    """
    Operators
    """

    # Arithmetic is straightforward
    # Testing using the assert function
    i1 = 1 
    i2 = 2 
    f1 = 1.0 
    f2 = 2.0 

    assert(i1 + i2 == 3)
    assert(i2 - i1 == 1)
    assert(i2 * i1 == 2)
    assert(i1 / i2 == 0)  # Floor/integer division 

    # You need to cast at least one integer to get a floating point result 
    # Casting is done with < and >
    x_cast1 = <float> i1 / i2
    printf("Should be 0.5: %f\n", x_cast1)  # 0.5
    printf("Should also be 0.5: %f\n", i1 / <double> i2)
    printf("Should also be 0.5: %f\n", f1 / f2)

    # Modulo exists as well
    assert(11 % 3 == 2)

    # Comparison operators return ints where 0 is trated as False 
    # and anything else is True
    assert((3 == 2) == 0)
    assert((3 != 2) == 1)
    assert(3 > 2)
    assert(not (3 < 2))
    assert(2 <= 2)
    assert(2 >= 2)

    # This is also not Python (yet...) - comparisons don't chain 
    # The line below evaluates to is treated as (7 > 6) > 5 
    # which simplifies to 1 > 5 which is False 
    a = 6 
    between_7_and_5 = 7 > a > 5
    assert(not between_7_and_5)

    # Instead chain using 'and'
    between_7_and_5 = 7 > a and a > 5
    assert(between_7_and_5)

    # Logic works on ints 
    assert(not 3 == 0)
    assert(not 0 == 1)
    assert(1 and 1)
    assert(not (0 and 1))
    assert(0 or 1)
    assert(not (0 or 0))

    # Increment and decrement operators
    j = 0 
    s = j++  # Return j THEN increase j
    assert(s == 0)
    assert(j == 1)

    s = --j  # Decrement j THEN return j 
    assert(s == 0)
    assert(j == 0)

    # Bitwise operators
    assert(~15 == -16)  # Bitwise negation/1's complement
    assert((2 & 3) == 2)  # Bitwise and 
    assert((2 | 3) == 3)  # Bitwise or
    assert((2 ^ 3) == 1)  # Bitwise xor 
    assert(1 << 1 == 2)  # Bitwise left shift 1
    assert(4 >> 1 == 2)  # Bitwise right shift 1 

    """
    Control flow
    """

    # If-elif-else ladder
    if 0:
        printf("I will never run\n")
    elif 0:
        printf("I will also never run\n")
    else:
        printf("This prints\n")

    # While loop
    ii = 0 
    while ii < 10:
        printf("%d, ", ii++)
    printf("\n")

    # Do-while loop 
    kk = 0 
    dowhile ++kk < 10:  # This gets executed after 1 cycle
        printf("%d, ", kk)
    printf("\n")

    # Switch statement 
    a = 3
    switch a:
        case 0:
            printf("a == 0\n")
            break 
        case 1:
            printf("a == 1\n")
            break 
        case 3, 4:
            printf("a is 3 or 4\n")
            break 
        else:
            fputs("Error\n", stderr)
            exit(-1)
            break 

    """
    Typecasting
    """

    # Cast to another type using < > (in C this is done with parenthesis)
    x_val = 1
    printf("%d\n", x_val)
    printf("%d\n", <short> x_val)
    printf("%d\n", <char> x_val)

    # Types will overflow without warning
    # char max == 255 if char is 8 bits long
    assert(<uchar> 257 == 1)

    # Integral types can be cast to floating point types and vice-versa
    printf("%f\n", <float>100)  # %f formats a float
    printf("%lf\n", <double>100)  # %lf formats a double
    printf("%d\n", <char>100.0)

    """
    Pointers
    """

    # A pointer is a variable declared to store a memory address. Its declaration will
    # also tell you the type of data it points to. You can retrieve the memory address
    # of your variables, then mess with them.

    x = 5
    printf("%p\n", <void[]>&x)  # Use & to retrieve the address of a variable 

    # Pointers are declared with empty brackets ([]),
    # similar to an array declaration, but without a size 
    px: int[] = &x
    not_a_pointer: int 
    printf("%p\n", <void[]> px)  # Print some address in memory
    printf("pointer size: %zu, int size: %zu\n", sizeof(px), sizeof(not_a_pointer))

    # To retrive the value at the address a pointer points to,
    # dereference it using '*'
    assert(*px == 5)

    # The pointer value can be changed 
    px_cpy = px
    (*px)++
    assert(*px == 6)
    assert(px == px_cpy)

    # Arrays allocate a continuous block of memory 
    x_array = [1, 2, 3, 4, 5]
    
    # Declare a pointer an initialize it to x_array 
    x_ptr = x_array 

    # x_ptr now points to the first element of x_array (1)
    assert(*x_ptr == 1)

    # Assign string 
    otherarr = "somestring"
    ptr = otherarr 
    printf("%s, %s\n", otherarr, ptr)

    # Pointers are incremented based on their type 
    assert(*(ptr + 1) == ptr[1])

    # You can dynamically allocate memory with malloc, which takes one 
    # argument of size_t representing the number of bytes to allocate.
    my_ptr = <int[]> malloc(sizeof(int) * 3)

    # Because malloc returns a void pointer, not specifying the type 
    # during initial assignment will cause my_ptr to be inferred as a 
    # void pointer. Casting it as an int pointer, or specifying the 
    # variable type creates a pointer of that type.
    # my_ptr: int[] = malloc(sizeof(int) * 3)  works also

    # Assign to malloc'd space
    my_ptr[0] = 1 
    my_ptr[1] = 2 
    my_ptr[2] = 3 
    assert(my_ptr[0] + my_ptr[1] == my_ptr[2])

    # Always remember to free malloc'd memory 
    free(my_ptr)

    # Call a function
    function_1()
    demo_func_ptrs()

    # End of main function 


# Function definition 
# Essentially the same as a declaration, but it has a body. 
# If the function type was declared before, the return type 
# and argument types match it, so it does not need to be 
# specifed again.
def add_two_ints(x1, x2):
    return x1 + x2 


# Function definition for a function that was not previously
# declared must specify argument types and return types, though
# if a return type is not specified, int is the default return type
def add_two_ints_plus_1(x1: int, x2: int) -> int:
    return x1 + x2 + 1


"""
User defined types and structs
"""

# Typedefs can be used to create type aliases 
typedef int my_type 
my_type_var: my_type = 0 

# Structs are collections of data where the members are allocated
# sequentially in the order they are written.
struct rectangle {
    width: uint,
    height: uint,
}


def function_1():
    # Newly defined structs are type'd as their name.
    # You do not type 'struct rectangle', just 'rectangle'
    my_rect: rectangle

    # Access struct members with '.'
    my_rect.width = 10 
    my_rect.height = 20 

    # Declare pointers to structs 
    my_rect_ptr = &my_rect 

    # Use derefencing to set sruct pointer members 
    (*my_rect_ptr).width = 30
    assert(my_rect.width == 30)

    # Alternatively use the -> shorthand for the sake of readability
    my_rect_ptr->width = 15
    assert(my_rect.width == 15)
    assert(my_rect_ptr->width == my_rect.width)


"""
Function pointers
"""

# Functions are also types and can be stored as variables like so 

def demo_func_ptrs():
    # Declare adder_func as a function which takes 2 ints and returns an int
    adder_func: (int, int) -> int
    adder_func = add_two_ints 

    # The type declaration above the assignment is unecessary since 
    # type inference would infer addr_func as the proper func type 
    # during assignment 
    assert(adder_func(1, 2) == 3)
    assert(adder_func(3, 0) == adder_func(1, 2))

    # New function assignment 
    adder_func = add_two_ints_plus_1
    assert(adder_func(1, 2) == 4)
